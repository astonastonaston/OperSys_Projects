﻿#include "virtual_memory.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ void init_invert_page_table(VirtualMemory *vm) {
  // [0,PE-1] valid bits and local count
  // [PE,2*PE-1] page number
  // [3*PE] global count
  // [2*PE, 3*PE-1] PID
  for (int i = 0; i < vm->PAGE_ENTRIES; i++) {
    vm->invert_page_table[i] = 0x80000000;
    vm->invert_page_table[i + vm->PAGE_ENTRIES] = i; // vpn -> ppn mapping 
    vm->invert_page_table[i + 2*vm->PAGE_ENTRIES] = -1; // process id
  }
  vm->invert_page_table[3*vm->PAGE_ENTRIES] = 0; // timer count
}

__device__ void vm_init(VirtualMemory *vm, uchar *buffer, uchar *storage,
                        u32 *invert_page_table, int *pagefault_num_ptr,
                        int PAGESIZE, int INVERT_PAGE_TABLE_SIZE,
                        int PHYSICAL_MEM_SIZE, int STORAGE_SIZE,
                        int PAGE_ENTRIES, int pid) {
  // init variables
  vm->pid = pid;
  vm->buffer = buffer;
  vm->storage = storage;
  vm->invert_page_table = invert_page_table;
  vm->pagefault_num_ptr = pagefault_num_ptr;

  // init constants
  vm->PAGESIZE = PAGESIZE;
  vm->INVERT_PAGE_TABLE_SIZE = INVERT_PAGE_TABLE_SIZE;
  vm->PHYSICAL_MEM_SIZE = PHYSICAL_MEM_SIZE;
  vm->STORAGE_SIZE = STORAGE_SIZE;
  vm->PAGE_ENTRIES = PAGE_ENTRIES;

  init_invert_page_table(vm);
}

__device__ uchar vm_read(VirtualMemory *vm, u32 addr) {
  /* Complate vm_read function to read single element from data buffer */
  for (int i = 0; i < vm->PAGE_ENTRIES; i++) {

    // valid block dealing
    if (!(vm->invert_page_table[i] & 0x80000000)) {
      // page hit
      if ((vm->invert_page_table[i + vm->PAGE_ENTRIES] == (addr/32)) && (vm->invert_page_table[i+2*vm->PAGE_ENTRIES] == threadIdx.x)) {
        // updata count
        vm->invert_page_table[i] &= 0xF0000000;
        vm->invert_page_table[i] += vm->invert_page_table[3*vm->PAGE_ENTRIES];
        vm->invert_page_table[3*vm->PAGE_ENTRIES] += 1;
	return vm->buffer[i*32+addr%32];
      }
      // page miss -> continue
    }

    // invalid block dealing
    else {
      // put disk page in memory
      for (int j = 0; j < vm->PAGESIZE; j++) {
        vm->buffer[i*32+j] = vm->storage[32*(addr/32)+j];
      }
      // update page table
      vm->invert_page_table[i + vm->PAGE_ENTRIES] = addr/32;
      vm->invert_page_table[i] &= 0x00000000;
      vm->invert_page_table[i+2*vm->PAGE_ENTRIES] = threadIdx.x;
      // updata count
      vm->invert_page_table[i] += vm->invert_page_table[3*vm->PAGE_ENTRIES];
      vm->invert_page_table[3*vm->PAGE_ENTRIES] = vm->invert_page_table[3*vm->PAGE_ENTRIES] + 1;
      // update page fault count
      *vm->pagefault_num_ptr = *vm->pagefault_num_ptr + 1;
      return vm->buffer[i*32+addr%32];
    }
  }

  // page fault and page table full
  int least_count = (1<<24), lruind, disk_pn;
  for (int i = 0; i < vm->PAGE_ENTRIES; i++) {
    if (((vm->invert_page_table[i]) & 0x0FFFFFFF) <= least_count) {
	lruind = i;
    	least_count = vm->invert_page_table[i] & 0x0FFFFFFF;
    }
  }
  disk_pn = vm->invert_page_table[lruind+vm->PAGE_ENTRIES];
  
  // write page back to disk
  // and update memory
  for (int i = 0; i < vm->PAGESIZE; i++) {
    vm->storage[32*disk_pn+i] = vm->buffer[lruind*32+i];
    vm->buffer[lruind*32+i] = vm->storage[32*(addr/32)+i];
  }

  // update current page table and count
  vm->invert_page_table[3*vm->PAGE_ENTRIES] = vm->invert_page_table[3*vm->PAGE_ENTRIES] + 1;
  vm->invert_page_table[lruind] &= 0x00000000;
  vm->invert_page_table[lruind] += vm->invert_page_table[3*vm->PAGE_ENTRIES];
  vm->invert_page_table[lruind+2*vm->PAGE_ENTRIES] = threadIdx.x;
  vm->invert_page_table[lruind + vm->PAGE_ENTRIES] = addr/32;

  // incr pgfault count
  *vm->pagefault_num_ptr = *vm->pagefault_num_ptr + 1;

  return vm->buffer[lruind*32+(addr%32)];
}



__device__ void vm_write(VirtualMemory *vm, u32 addr, uchar value) {
  // scanning for page hit cases
  for (int i = 0; i < vm->PAGE_ENTRIES; i++) {

    // valid block dealing
    if (!(vm->invert_page_table[i] & 0x80000000)) {
      // page hit
      if ((vm->invert_page_table[i + vm->PAGE_ENTRIES] == (addr/32)) && (vm->invert_page_table[i+2*vm->PAGE_ENTRIES] == threadIdx.x)) {
        // write to memory buffer
        vm->buffer[i*32+addr%32] = value;

        // updata count
        vm->invert_page_table[i] &= 0xF0000000;
        vm->invert_page_table[i] += vm->invert_page_table[3*vm->PAGE_ENTRIES];
        vm->invert_page_table[3*vm->PAGE_ENTRIES] += 1;
        return ;
      }
      // page miss -> continue
    }

    // invalid block dealing
    else {
      // put disk page in memory
      for (int j = 0; j < vm->PAGESIZE; j++) {
        vm->buffer[i*32+j] = vm->storage[32*(addr/32)+j];
      }
      // write to memory buffer
      vm->buffer[i*32+addr%32] = value;

      // update page table vpn and dirty&invalid bit 
      // and count
      vm->invert_page_table[i + vm->PAGE_ENTRIES] = addr/32;

      vm->invert_page_table[i] &= 0x00000000;
      vm->invert_page_table[i] += vm->invert_page_table[3*vm->PAGE_ENTRIES];
      vm->invert_page_table[i+2*vm->PAGE_ENTRIES] = threadIdx.x;
      vm->invert_page_table[3*vm->PAGE_ENTRIES] = vm->invert_page_table[3*vm->PAGE_ENTRIES] + 1;
      // update page fault count
      *vm->pagefault_num_ptr = *vm->pagefault_num_ptr + 1;

      return ;
    }
  }  

  // page fault and page table full
  int least_count = (1<<24), lruind, disk_pn;
  for (int i = 0; i < vm->PAGE_ENTRIES; i++) {
    if ((vm->invert_page_table[i] & 0x0FFFFFFF) <= least_count) {
	lruind = i;
    	least_count = vm->invert_page_table[i] & 0x0FFFFFFF;
    }
  }
  disk_pn = vm->invert_page_table[lruind+vm->PAGE_ENTRIES];
  
  // write page back to disk
  // and update memory
  for (int i = 0; i < vm->PAGESIZE; i++) {
    vm->storage[32*disk_pn+i] = vm->buffer[lruind*32+i];
    vm->buffer[lruind*32+i] = vm->storage[32*(addr/32)+i];
  }
  vm->buffer[lruind*32+(addr%32)] = value;
  
  // update current page table and count
  vm->invert_page_table[lruind] &= 0x00000000;
  vm->invert_page_table[lruind] += vm->invert_page_table[3*vm->PAGE_ENTRIES];
  vm->invert_page_table[lruind + vm->PAGE_ENTRIES] = addr/32; 
  vm->invert_page_table[lruind+2*vm->PAGE_ENTRIES] = threadIdx.x;
  vm->invert_page_table[3*vm->PAGE_ENTRIES] = vm->invert_page_table[3*vm->PAGE_ENTRIES] + 1;
  
  // incr pgfault count
  *vm->pagefault_num_ptr = *vm->pagefault_num_ptr + 1;
  
  return ;
}

__device__ void vm_snapshot(VirtualMemory *vm, uchar *results, int offset,
                            int input_size) {
  for (int i=offset; i<input_size; i++) {
    results[i] = vm_read(vm, i);
  }
}

